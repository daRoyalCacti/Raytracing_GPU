#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "common.h"

#include "hittable_list.h"

//for compiling purposes
#include "color.h"
#include "camera.h"
#include "moving_sphere.h"
#include "aarect.h"
#include "box.h"
#include "constant_medium.h"
#include "bvh.h"

#include "scenes.h"


__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {	//no need for parallism
		*(d_list)   = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0, 1, 0)));
		*(d_list+1) = new sphere(vec3(0,-100.5,-1), 100, new lambertian(vec3(0, 0, 1)));
		*d_world    = new hittable_list(d_list,2);
		*d_camera   = new camera(vec3(0,0,-3), vec3(0,0,0), vec3(0,1,0), 40, 16.0f/9.0f, 0.0f, 10.0f, 0, 1 );
	}
}





__device__ vec3 color_f(ray& r, hittable **world, hiprandState *local_rand_state, int depth) {
	const vec3 background(0.7f, 0.8f, 1.0f);

	hit_record rec;

	if (depth <= 0)
		return color(0,0,0);
	
	if (!(*world)->hit(r, 0.001f, infinity, rec, local_rand_state)) 
		return background;

	ray scattered;
	color attenuation;
	const color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);

	if (!rec.mat_ptr->scatter(r, rec, attenuation, scattered, local_rand_state))
		return emitted;
	
	return emitted + attenuation*color_f(scattered, world, local_rand_state, depth-1);	
}


__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j*max_x + i;
	//Initialising for random numbers
	//Each thread gets the same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera **cam, hiprandState *rand_state,  hittable **world, int max_depth) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;	//if trying the work with more values than wanted
	int pixel_index = j*max_x + i;

	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0,0,0);
	
	for(int s=0; s < ns; s++) {
		float u = float(i+random_float(&local_rand_state)) / max_x;
		float v = float(j+random_float(&local_rand_state)) / max_y;
		
		ray r = (*cam)->get_ray(rand_state, u,v);
		col += color_f(r, world, &local_rand_state, max_depth);
	}

	fb[pixel_index] = col/float(ns);
}



int main() {
	const unsigned nx = 1200;	//image width in frame buffer (also the output image size)
	const double aspect_ratio = 16.0 / 9.0;
	const unsigned ny = static_cast<unsigned>(nx / aspect_ratio);
	const unsigned num_pixels = nx*ny;
	const unsigned ns = 100;	//rays per pixel

	const unsigned tx = 8;	//dividing the work on the GPU into
	const unsigned ty = 8; 	//threads of tx*ty threads

	std::cerr << "Generating a " << nx << "x" << ny << " image with " << ns << " rays per pixel\n";
	std::cerr << "using " << tx << "x" << ty << " blocks.\n";



	std::cerr << "Allocating Frame Buffer" << std::flush;
	//Frame buffer (holds the image in the GPU)
	vec3 *fb;
	const size_t fb_size = num_pixels*sizeof(vec3);	
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));	//allocating the frame buffer on the GPU
	
	std::cerr << "\rCreating World" << std::flush;
	//scene curr_scene = basic_scene(aspect_ratio);
	//make our world of hittables and the camera
	hittable **d_list;
	checkCudaErrors(hipMalloc((void**)&d_list, 2*sizeof(hittable*) ));	//2 because 2 hittables
	hittable **d_world;
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable *) ));

	camera **d_camera;
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*) ));

	create_world<<<1,1>>>(d_list, d_world, d_camera);		//create_world is defined above
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());	//tell cpu the world is created
	
	
	//Render to the frame buffer
	dim3 blocks(nx/tx+1, ny/ty+1);
	dim3 threads(tx, ty);
	hiprandState *d_rand_state;
	checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels*sizeof(hiprandState) ));

	std::cerr << "\rIntialising the render" << std::flush;
	render_init<<<blocks, threads>>>(nx, ny, d_rand_state);		//initialising the render -- currently just setting up the random numbers
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	
	std::cerr << "\rRendering to frame buffer" << std::flush;
	render<<<blocks, threads>>>(fb, nx, ny, ns,	//render is a function defined above
					d_camera,
					d_rand_state,
					d_world, 10);		
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());	//tells the CPU that the GPU is done rendering
	
	std::cerr << "\rOutputting image" << std::flush;
	write_frame_buffer(std::cout, fb, nx, ny);

	std::cerr << "\rCleaning Up" << std::flush;	
	//clean up
	checkCudaErrors(hipDeviceSynchronize());
	free_world<<<1,1>>>(d_list,d_world,d_camera,2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(fb));

	std::cerr << std::endl;

	return 0;
}
